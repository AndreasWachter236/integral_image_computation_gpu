#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include "image.h"

const int blockSize = 1024;
const int gridSize = 64;

int const image_width = 10000;
int const image_height = 100000;

using namespace std;

template <typename T>
void createRandomImage(Image<T>& rndImage)
{
    for (int j = 0; j < rndImage.pixels; j++)
    {
        rndImage.data[j] = 1;// (T) rand() % 256);
    }
}


template <typename T>
T sum(Image<T>& iImage, const size_t x, const size_t y, const size_t w, const size_t h)
{
    T a = iImage.get(x, y);
    T b = iImage.get(x + w, y);
    T c = iImage.get(x, y + h);
    T d = iImage.get(x + w, y + h);

    return d - b - c + a;
}

template <typename T>
__global__ void calculate_iImage(T* iImage,const size_t width, const size_t height)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < height)
    {
        for (size_t i = 1; i < width; i++)
        {
            //iImage[j * width + i] = iImage[j * width + i] + iImage[j * width + i-1];
            atomicAdd(&(iImage[j * width + i]), iImage[j * width + i - 1]);
        }
    }

    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width)
    {
        for (size_t j = 1; j < height; j++)
        {
            atomicAdd(&(iImage[j * width + i]), iImage[(j - 1) * width + i]);
            //iImage[j * width + i] = iImage[j * width + i] + iImage[(j-1) * width + i];
        }
    }
}

int main()
{
    auto tStart = chrono::high_resolution_clock::now();
    Image<int> iImage_cpu;
    iImage_cpu.create(image_width, image_height);

    // allocate memory on cpu //
    hipHostMalloc((void**)&iImage_cpu.data, iImage_cpu.pixels * sizeof(int));
    createRandomImage(iImage_cpu);

    // allocate memory on gpu //
    Image<int> iImage_gpu;
    iImage_gpu.create(image_width, image_height);
    std::copy(&iImage_cpu.data[0], &iImage_cpu.data[0] + iImage_cpu.pixels, &iImage_gpu.data[0]);

    hipMalloc((void**) & iImage_gpu.data, iImage_gpu.pixels * sizeof(int));

    // copy data to gpu //
    hipMemcpy(iImage_gpu.data, iImage_cpu.data, iImage_gpu.pixels * sizeof(int), hipMemcpyHostToDevice);

    // run calculation //
    calculate_iImage <<< gridSize, blockSize >>>(iImage_gpu.data, iImage_gpu.width, iImage_gpu.height);

    hipMemcpy(iImage_cpu.data, iImage_gpu.data, iImage_gpu.pixels * sizeof(int), hipMemcpyDeviceToHost);

    auto tEnd = chrono::high_resolution_clock::now();
    auto runtime = chrono::duration_cast<chrono::nanoseconds>(tEnd - tStart);

    printf("Time measured: %.3f seconds.\n", runtime.count() * 1e-9);

    //cout << iImage_cpu.get(999,999) << endl;
    // free gpu memory //
    hipFree(iImage_gpu.data);

    // free cpu memory //
    hipHostFree(iImage_cpu.data);

    return 0;
}
